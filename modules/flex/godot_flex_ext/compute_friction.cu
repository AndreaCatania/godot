#include "hip/hip_runtime.h"
/*************************************************************************/
/*  compute_friction.cu                                                  */
/*************************************************************************/
/*                       This file is part of:                           */
/*                           GODOT ENGINE                                */
/*                      https://godotengine.org                          */
/*************************************************************************/
/* Copyright (c) 2007-2018 Juan Linietsky, Ariel Manzur.                 */
/* Copyright (c) 2014-2018 Godot Engine contributors (cf. AUTHORS.md)    */
/*                                                                       */
/* Permission is hereby granted, free of charge, to any person obtaining */
/* a copy of this software and associated documentation files (the       */
/* "Software"), to deal in the Software without restriction, including   */
/* without limitation the rights to use, copy, modify, merge, publish,   */
/* distribute, sublicense, and/or sell copies of the Software, and to    */
/* permit persons to whom the Software is furnished to do so, subject to */
/* the following conditions:                                             */
/*                                                                       */
/* The above copyright notice and this permission notice shall be        */
/* included in all copies or substantial portions of the Software.       */
/*                                                                       */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,       */
/* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF    */
/* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.*/
/* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY  */
/* CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,  */
/* TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE     */
/* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.                */
/*************************************************************************/

/**
	@author AndreaCatania
*/

/**
 * This is writtein in Cuda C
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "thirdparty/flex/include/NvFlex.h"
#include "thirdparty/flex/core/maths.h"

static const int kNumThreadsPerBlock = 256;

struct GdFlexExtComputeFrictionCallback{

	NvFlexSolver *solver;

	int primitive_body_count;

	// Previous frame primitive transform
	float* primitive_prev_transfs;
	float* primitive_inv_prev_transfs;

	// Current invers transform
	float* primitive_inv_curr_transfs;

	// Motion that primitive did to reach current transform
	float* primitive_motion_transfs;

	//float* primitive_aabbs;

	float* primitive_extent;
	float* primitive_frictions;
	float* primitive_friction_2_thresholds;
	uint32_t* primitive_layers;
	float primitive_margin;

	int particle_count;
	float* prev_particles_position_mass;

	float particle_radius;


	GdFlexExtComputeFrictionCallback(NvFlexSolver* p_solver)
		: solver(p_solver),
		  primitive_body_count(0),
		  primitive_prev_transfs(NULL),
		  primitive_inv_prev_transfs(NULL),
		  primitive_inv_curr_transfs(NULL),
		  primitive_motion_transfs(NULL),
		  //primitive_aabbs(NULL),
		  primitive_extent(NULL),
		  primitive_frictions(NULL),
		  primitive_friction_2_thresholds(NULL),
		  primitive_layers(NULL),
		  primitive_margin(0.01),
		  particle_count(0),
		  prev_particles_position_mass(NULL),
		  particle_radius(0.1)
	{}

	~GdFlexExtComputeFrictionCallback(){
		free_primitives();
		free_particles();
	}

	void free_primitives(){
		hipFree(primitive_prev_transfs);
		hipFree(primitive_inv_prev_transfs);
		hipFree(primitive_inv_curr_transfs);
		hipFree(primitive_motion_transfs);
		//hipFree(primitive_aabbs);
		hipFree(primitive_extent);
		hipFree(primitive_frictions);
		hipFree(primitive_friction_2_thresholds);
		hipFree(primitive_layers);

		primitive_prev_transfs = NULL;
		primitive_inv_prev_transfs = NULL;
		primitive_inv_curr_transfs = NULL;
		primitive_motion_transfs = NULL;
		//primitive_aabbs = NULL;
		primitive_extent = NULL;
		primitive_frictions = NULL;
		primitive_friction_2_thresholds = NULL;
		primitive_layers = NULL;

		primitive_body_count = 0;
	}

	void free_particles(){
		hipFree(prev_particles_position_mass);
		prev_particles_position_mass = NULL;

		particle_count = 0;
	}
};

GdFlexExtComputeFrictionCallback *GdFlexExtCreateComputeFrictionCallback(NvFlexSolver *p_solver){
	return new GdFlexExtComputeFrictionCallback(p_solver);
}

void GdFlexExtDestroyComputeFrictionCallback(GdFlexExtComputeFrictionCallback* p_callback){
	delete p_callback;
}

struct GdTransform{
	Matrix33 basis;
	Vec3 origin;


	__device__ Vec3 xform(const Vec3& other) const {

		return Vec3(
					Dot3(basis.cols[0], other) + origin.x,
					Dot3(basis.cols[1], other) + origin.y,
					Dot3(basis.cols[2], other) + origin.z);
	}

	/// The last component of vector is ignored
	__device__ Vec3 xform(const Vec4& other) const {

		return Vec3(
					Dot3(basis.cols[0], other) + origin.x,
					Dot3(basis.cols[1], other) + origin.y,
					Dot3(basis.cols[2], other) + origin.z);
	}

	__device__ void translate(const Vec3& vec){
		origin.x += Dot3(basis.cols[0], vec);
		origin.y += Dot3(basis.cols[1], vec);
		origin.z += Dot3(basis.cols[2], vec);
	}

	__device__ GdTransform translated(const Vec3& vec) const {
		GdTransform t = *this;
		t.translate(vec);
		return t;
	}

	__device__ void operator*=(const GdTransform &p_transform) {

		origin = xform(p_transform.origin);
		basis = Multiply(basis, p_transform.basis);
	}

	__device__ GdTransform operator*(const GdTransform &p_transform) const {

		GdTransform t = *this;
		t *= p_transform;
		return t;
	}
};

__device__ float get_sphere_penetration(
		const Vec3 &box_half_extent,
		const Vec3 &sphere_rel_pos,
		Vec3 &r_closest_point,
		Vec3 &r_normal ){

	//project the center of the sphere on the closest face of the box
	float face_dist = box_half_extent.x - sphere_rel_pos.x;
	float min_dist = face_dist;
	r_closest_point = sphere_rel_pos;
	r_closest_point.x = box_half_extent.x;
	r_normal.x = 1.0;
	r_normal.y = 0.0;
	r_normal.z = 0.0;

	face_dist = box_half_extent.x + sphere_rel_pos.x;
	if (face_dist < min_dist){
		min_dist = face_dist;
		r_closest_point = sphere_rel_pos;
		r_closest_point.x = -box_half_extent.x;
		r_normal.x = -1.0;
		r_normal.y = 0.0;
		r_normal.z = 0.0;
	}

	face_dist = box_half_extent.y - sphere_rel_pos.y;
	if (face_dist < min_dist)
	{
		min_dist = face_dist;
		r_closest_point = sphere_rel_pos;
		r_closest_point.y = box_half_extent.y;
		r_normal.x = 0.0;
		r_normal.y = 1.0;
		r_normal.z = 0.0;
	}

	face_dist = box_half_extent.y + sphere_rel_pos.y;
	if (face_dist < min_dist)
	{
		min_dist = face_dist;
		r_closest_point = sphere_rel_pos;
		r_closest_point.y = -box_half_extent.y;
		r_normal.x = 0.0;
		r_normal.y = -1.0;
		r_normal.z = 0.0;
	}

	face_dist = box_half_extent.z - sphere_rel_pos.z;
	if (face_dist < min_dist)
	{
		min_dist = face_dist;
		r_closest_point = sphere_rel_pos;
		r_closest_point.z = box_half_extent.z;
		r_normal.x = 0.0;
		r_normal.y = 0.0;
		r_normal.z = 1.0;
	}

	face_dist = box_half_extent.z + sphere_rel_pos.z;
	if (face_dist < min_dist)
	{
		min_dist = face_dist;
		r_closest_point = sphere_rel_pos;
		r_closest_point.z = -box_half_extent.z;
		r_normal.x = 0.0;
		r_normal.y = 0.0;
		r_normal.z = -1.0;
	}

	return min_dist;
}

__device__ float length_2(const Vec3 &vec){

	return
			vec.x * vec.x +
			vec.y * vec.y +
			vec.z * vec.z;
}

__device__ bool compute_collision_box(const Vec4& particle_pos,
									  const GdTransform* p_primitive_inv_transf,
									  const Vec3* extent,
									  float radius,
									  float margin,
									  float *r_penetration_dist = NULL,
									  Vec3 *r_normal = NULL,
									  Vec3 *r_closest_point_on_box = NULL){


	Vec3 local_ppos = p_primitive_inv_transf->xform(particle_pos);

	Vec3 closest_point(local_ppos);

	closest_point.x = Min(closest_point.x, extent->x);
	closest_point.x = Max(closest_point.x, -extent->x);

	closest_point.y = Min(closest_point.y, extent->y);
	closest_point.y = Max(closest_point.y, -extent->y);

	closest_point.z = Min(closest_point.z, extent->z);
	closest_point.z = Max(closest_point.z, -extent->z);

	Vec3 collision_point = local_ppos - closest_point;

	const float squared_dist = length_2(collision_point);

	const bool collision = squared_dist < radius * radius + margin;

	if(collision){

		if(r_penetration_dist && r_normal && r_closest_point_on_box){

			Vec3 normal(0, 0, 0);
			if(squared_dist <= FLT_EPSILON){
				// When the distance is lower than epsilon the sphere center is
				// inside the box
				(*r_penetration_dist) =
						-get_sphere_penetration(
							*extent,
							local_ppos,
							closest_point,
							normal);
				normal *= -1;

			}else{
				float dist = sqrt(squared_dist);
				(*r_penetration_dist) = dist - radius;

				if (dist != 0) {
					normal = collision_point / dist;
				}
			}

			(*r_closest_point_on_box) = closest_point;
			(*r_normal) = normal;
		}

	}

	return collision;
}

__device__ bool AABB_intersect(Vec3 aabb1_min,
							   Vec3 aabb1_max,
							   Vec3 aabb2_min,
							   Vec3 aabb2_max) {

	if(aabb1_min.x >= aabb2_max.x)
		return false;

	if(aabb1_max.x <= aabb2_min.x)
		return false;

	if(aabb1_min.y >= aabb2_max.y)
		return false;

	if(aabb1_max.y <= aabb2_min.y)
		return false;

	if(aabb1_min.z >= aabb2_max.z)
		return false;

	if(aabb1_max.z <= aabb2_min.z)
		return false;

	return true;
}

__device__ Vec3 get_contact_velocity(
		float inverse_dt,
		const Vec4 &f,
		const Vec4 &t,
		float friction){

	const float alpha = 1 - friction;

	return Vec3(
			(alpha * (t.x - f.x)) * inverse_dt,
			(alpha * (t.y - f.y)) * inverse_dt,
			(alpha * (t.z - f.z)) * inverse_dt);
}

__device__ Vec4 linear_interpolation(
		const Vec4 &f,
		const Vec4 &t,
		float alpha){

	return Vec4(
			f.x + (alpha * (t.x - f.x)),
			f.y + (alpha * (t.y - f.y)),
			f.z + (alpha * (t.z - f.z)),
			f.w);
}

__global__ void ComputeFriction(
		const float p_inverse_dt,
		const int *p_sorted_to_original,
		Vec4* __restrict__ r_particle_positions,
		Vec4* __restrict__ r_particle_velocities,
		const int* p_particle_phases,
		const int p_primitive_body_count,
		const GdTransform *p_primitive_prev_transfs,
		const GdTransform *p_primitive_inv_prev_transfs,
		const GdTransform *p_primitive_inv_curr_transfs,
		const GdTransform *p_primitive_motion_transfs,
		//const Vec3 *p_primitive_aabbs,
		const Vec3 *p_primitive_extents,
		const float *p_primitive_frictions,
		const float *p_primitive_friction_2_thresholds,
		const uint32_t* p_primitive_layers,
		const float p_primitive_margin,
		const Vec4* p_prev_particles_position_mass,
		const float p_particle_radius ){

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int sorted_particle_index = i;

	const int original_particle_index = p_sorted_to_original[sorted_particle_index];

	const Vec4 curr_particle_pos = r_particle_positions[sorted_particle_index];
	const int phase = p_particle_phases[sorted_particle_index];
	const int particle_layer = phase >> 24;

	const Vec4 prev_particle_pos = p_prev_particles_position_mass[original_particle_index];

	//TEST_AABB
	//const Vec3 radius_vec(radius, radius, radius);
	//const Vec3 current_p_pos_vec3 = Vec3(current_p_pos.x, current_p_pos.y, current_p_pos.z);
	//
	//Vec3 particle_aabb_begin = current_p_pos - radius_vec;
	//Vec3 particle_aabb_end = current_p_pos + radius;

	for(int p = 0; p < p_primitive_body_count; p++ ){

		const float friction = p_primitive_frictions[p];
		const float friction_threshold = p_primitive_friction_2_thresholds[p];
		const uint32_t layer = p_primitive_layers[p];

		if(!(layer & particle_layer))
			continue;

		//if( !AABB_intersect(
		//		particle_aabb_begin,
		//		particle_aabb_end,
		//		p_primitive_aabbs[p*2+0],
		//		p_primitive_aabbs[p*2+0] + p_primitive_aabbs[p*2+1]) )
		//	continue;

		/// NOTE
		/// To make the calculation more lightweight the two cases when
		/// the particle enter or exit are not properly handled.
		/// This mean that when the particle exit the surface this algorithm
		/// doesn't compute the friction.
		/// However this is not a big problem thanks to high frame rate

		/// Step 1
		/// Check if the particle collide or is escaped from the rough surface



		if( !compute_collision_box(
					curr_particle_pos,
					p_primitive_inv_curr_transfs + p,
					p_primitive_extents + p,
					p_particle_radius,
					p_primitive_margin) )
			continue;

		/// Step 2
		/// Check if previously the particle was colliding
		///
		/// This can't be cached due to the fact that the particles ID
		/// can change frame by frame. (More work should be required)

		Vec3 normal;
		Vec3 closest_point_on_box;
		float penetration;

		if( !compute_collision_box(
					prev_particle_pos,
					p_primitive_inv_prev_transfs + p,
					p_primitive_extents + p,
					p_particle_radius,
					p_primitive_margin,
					&penetration,
					&normal,
					&closest_point_on_box) )
			continue;

		/// Step 3
		/// Compute motion of primitive body with particle
		/// attached to the safe position to prevent overlapping

		Vec3 safe_relative_prev_particle_pos =
				closest_point_on_box + normal * p_particle_radius;
				//closest_point_on_box + normal * (p_particle_radius + p_primitive_margin);

		GdTransform full_friction_particle_trs =
				p_primitive_prev_transfs[p].translated(safe_relative_prev_particle_pos) *
				p_primitive_motion_transfs[p];

		/// Step 4
		/// Calculate new position and velocity according to
		/// static and and dynamic friction

		Vec3 delta_movement(
					curr_particle_pos.x - full_friction_particle_trs.origin.x,
					curr_particle_pos.y - full_friction_particle_trs.origin.y,
					curr_particle_pos.z - full_friction_particle_trs.origin.z);

		float alpha = 0; // Static friction

		if(friction_threshold < length_2(delta_movement))
			alpha = 1 - friction; // Dynamic friction

		delta_movement *= alpha;

		const Vec4 new_p_pos(full_friction_particle_trs.origin + delta_movement, curr_particle_pos.w);

		r_particle_positions[sorted_particle_index] = new_p_pos;

		r_particle_velocities[sorted_particle_index] =
				(new_p_pos - prev_particle_pos) * p_inverse_dt;
		// last (W) is not used but reset it
		r_particle_velocities[sorted_particle_index][3] = 0;

		// TODO in case of multi contact this is no more correct
		// For this reason allow only one contact
		return;
	}
}

void ComputeFrictionCallback(NvFlexSolverCallbackParams p_params){

	GdFlexExtComputeFrictionCallback* callback =
			static_cast<GdFlexExtComputeFrictionCallback*>(p_params.userData);

	const float inverse_dt = 1.0 / p_params.dt;
	const int particle_count = p_params.numActive;
	const int kNumBlocks = (particle_count + kNumThreadsPerBlock - 1) / kNumThreadsPerBlock;

	ComputeFriction<<<kNumBlocks, kNumThreadsPerBlock>>>(
			inverse_dt,
			p_params.sortedToOriginalMap,
			(Vec4*)p_params.particles,
			(Vec4*)p_params.velocities,
			p_params.phases,
			callback->primitive_body_count,
			(GdTransform*)callback->primitive_prev_transfs,
			(GdTransform*)callback->primitive_inv_prev_transfs,
			(GdTransform*)callback->primitive_inv_curr_transfs,
			(GdTransform*)callback->primitive_motion_transfs,
			//(Vec3*)callback->primitive_aabbs,
			(Vec3*)callback->primitive_extent,
			callback->primitive_frictions,
			callback->primitive_friction_2_thresholds,
			callback->primitive_layers,
			callback->primitive_margin,
			(Vec4*)callback->prev_particles_position_mass,
			callback->particle_radius);
}

void GdFlexExtSetComputeFrictionCallback(
		GdFlexExtComputeFrictionCallback* p_callback,
		const int p_primitive_body_count,
		const float *p_primitive_prev_transfs,
		const float *p_primitive_inv_prev_transfs,
		const float *p_primitive_inv_curr_transfs,
		const float *p_primitive_motions,
		//const float *p_primitive_aabbs,
		const float *p_primitive_extents,
		const float *p_primitive_frictions,
		const float *p_primitive_friction_2_thresholds,
		const uint32_t * p_primitive_layers,
		const float p_primitive_margin,
		const int p_particle_count,
		const float *p_prev_particles_position_mass,
		const float p_particle_radius){

	if( p_callback->primitive_body_count != p_primitive_body_count ){

		p_callback->free_primitives();

		p_callback->primitive_body_count = p_primitive_body_count;

		if(p_primitive_body_count){
			hipMalloc(&p_callback->primitive_prev_transfs,
					   sizeof(float) * p_primitive_body_count * 12); // Transform

			hipMalloc(&p_callback->primitive_inv_prev_transfs,
					   sizeof(float) * p_primitive_body_count * 12); // Transform

			hipMalloc(&p_callback->primitive_inv_curr_transfs,
					   sizeof(float) * p_primitive_body_count * 12); // Transform

			hipMalloc(&p_callback->primitive_motion_transfs,
					   sizeof(float) * p_primitive_body_count * 12); // Transform

			//hipMalloc(&p_callback->primitive_aabbs,
			//		   sizeof(float) * p_primitive_body_count * 6); // AABB

			hipMalloc(&p_callback->primitive_extent,
					   sizeof(float) * p_primitive_body_count * 3); // Vector3

			hipMalloc(&p_callback->primitive_frictions,
					   sizeof(float) * p_primitive_body_count * 1); // Float

			hipMalloc(&p_callback->primitive_friction_2_thresholds,
					   sizeof(float) * p_primitive_body_count * 1); // Float

			hipMalloc(&p_callback->primitive_layers,
					   sizeof(uint32_t) * p_primitive_body_count * 1); // unsigned int
		}
	}

	if(p_callback->particle_count != p_particle_count){

		p_callback->free_particles();
		p_callback->particle_count = p_particle_count;

		if(p_particle_count){
			hipMalloc(
					&p_callback->prev_particles_position_mass,
					sizeof(float) * p_particle_count * 4 ); // Vector4
		}
	}

	if(!p_primitive_body_count)
		return;

	if(!p_particle_count)
		return;

	hipMemcpy(p_callback->primitive_prev_transfs,
			   p_primitive_prev_transfs,
			   sizeof(float) * p_primitive_body_count * 12,
			   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->primitive_inv_prev_transfs,
			   p_primitive_inv_prev_transfs,
			   sizeof(float) * p_primitive_body_count * 12,
			   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->primitive_inv_curr_transfs,
			   p_primitive_inv_curr_transfs,
			   sizeof(float) * p_primitive_body_count * 12,
			   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->primitive_motion_transfs,
			   p_primitive_motions,
			   sizeof(float) * p_primitive_body_count * 12,
			   hipMemcpyHostToDevice);

	//hipMemcpy(p_callback->d_primitive_aabbs,
	//		   p_primitive_aabbs,
	//		   sizeof(float) * p_primitive_body_count * 6,
	//		   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->primitive_extent,
			   p_primitive_extents,
			   sizeof(float) * p_primitive_body_count * 3,
			   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->primitive_frictions,
			   p_primitive_frictions,
			   sizeof(float) * p_primitive_body_count * 1,
			   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->primitive_friction_2_thresholds,
			   p_primitive_friction_2_thresholds,
			   sizeof(float) * p_primitive_body_count * 1,
			   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->primitive_layers,
			   p_primitive_layers,
			   sizeof(uint32_t) * p_primitive_body_count * 1,
			   hipMemcpyHostToDevice);

	hipMemcpy(p_callback->prev_particles_position_mass,
			   p_prev_particles_position_mass,
			   sizeof(float) * p_particle_count * 4,
			   hipMemcpyHostToDevice);


	p_callback->primitive_margin = p_primitive_margin;
	p_callback->particle_radius = p_particle_radius;

	NvFlexSolverCallback solver_callback;
	solver_callback.function = ComputeFrictionCallback;
	solver_callback.userData = p_callback;

	NvFlexRegisterSolverCallback(p_callback->solver,
								 solver_callback,
								 eNvFlexStageSubstepEnd);
}
